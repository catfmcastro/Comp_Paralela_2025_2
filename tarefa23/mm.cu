// 


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err) { \
  if(err != hipSuccess) { \
    fprintf(stderr, "Erro CUDA: %s em %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
    exit(EXIT_FAILURE); \
  } \
}

// kernel
__global__ void mm_kernel(double* d_a, double* d_b, double* d_c, int width) 
{
    // Calcula o índice global da thread na matriz (linha e coluna)
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // Checagem de limites: garante que a thread está dentro dos limites da matriz
    if (row < width && col < width) {
        double sum = 0;
        
        // Loop 'k' original para calcular o produto escalar
        for (int k = 0; k < width; k++) {
            double x = d_a[row * width + k]; // Elemento da linha 'row' de 'a'
            double y = d_b[k * width + col]; // Elemento da coluna 'col' de 'b'
            sum += x * y;
        }
        
        // Armazena o resultado na posição correta em 'c'
        d_c[row * width + col] = sum;
    }
}

// função host
int main()
{
    int width = 2000;
    size_t size = (size_t)width * width * sizeof(double);

    // alocar memória no host (CPU)
    double *a = (double*) malloc (size);
    double *b = (double*) malloc (size);
    double *c = (double*) malloc (size);

    // inicializar dados no host
    for(int i = 0; i < width; i++) {
        for(int j = 0; j < width; j++) {
            a[i*width+j] = (double)i;
            b[i*width+j] = (double)j;
            c[i*width+j] = 0;
        }
    }

    // alocar memória no devide (GPU)
    double *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc((void**) &d_a, size));
    CUDA_CHECK(hipMalloc((void**) &d_b, size));
    CUDA_CHECK(hipMalloc((void**) &d_c, size));

    // copiar dados do host para o device
    CUDA_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));

    // tamanho do bloco de threads
    dim3 threadsPerBlock(16, 16);
    
    // define o tamanho do grid
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (width + threadsPerBlock.y - 1) / threadsPerBlock.y);

    
    // inicia o kernel
    mm_kernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, width);
    
    // sincroniza
    CUDA_CHECK(hipDeviceSynchronize());

    // copia resultados do device para o host
    CUDA_CHECK(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

    free(a);
    free(b);
    free(c);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}